
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#define SIZE	1024

__global__ void VectorAdd(float *a, float *b, float *c, int n)
{
	int i = threadIdx.x;

	if (i < n)
		c[i] = a[i] + b[i];
}

int main()
{
	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	clock_t start, end;
	double cpu_time_used;

	a = (float *)malloc(SIZE*sizeof(float));
	b = (float *)malloc(SIZE*sizeof(float));
	c = (float *)malloc(SIZE*sizeof(float));

	hipMalloc( &d_a, SIZE*sizeof(float));
	hipMalloc( &d_b, SIZE*sizeof(float));
	hipMalloc( &d_c, SIZE*sizeof(float));

	for( int i = 0; i < SIZE; ++i )
	{
		a[i] = (float) i;
		b[i] = (float) i;
		c[i] = 0.0;
	}

	hipMemcpy( d_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, SIZE*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, SIZE*sizeof(float), hipMemcpyHostToDevice );

	start = clock();
	VectorAdd<<< 1, SIZE >>>(d_a, d_b, d_c, SIZE);
	end = clock();

	hipMemcpy( c, d_c, SIZE*sizeof(float), hipMemcpyDeviceToHost );

	for( int i = 0; i < 10; ++i)
		printf("c[%d] = %f\n", i, c[i]);

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	cpu_time_used = ((double) (end - start))/CLOCKS_PER_SEC;
	printf("Time = %f seconds to execute.\n", cpu_time_used);
	return 0;
}
